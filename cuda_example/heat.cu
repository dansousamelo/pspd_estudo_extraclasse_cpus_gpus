#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <iomanip>

#define N 1024   // Tamanho da placa (N x N)
#define T 1000   // Número de iterações
#define BLOCK_SIZE 32  // Tamanho do bloco CUDA

__global__ void heat_kernel(float* d_plate, float* d_new_plate, int n) {
    __shared__ float shared_plate[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int local_x = threadIdx.x + 1;
    int local_y = threadIdx.y + 1;

    if (x < n && y < n) {
        // Carregar a célula principal e vizinhos na memória compartilhada
        shared_plate[local_x][local_y] = d_plate[y * n + x];

        if (threadIdx.x == 0 && x > 0) // Carregar vizinho esquerdo
            shared_plate[0][local_y] = d_plate[y * n + x - 1];
        if (threadIdx.x == BLOCK_SIZE - 1 && x < n - 1) // Vizinho direito
            shared_plate[BLOCK_SIZE + 1][local_y] = d_plate[y * n + x + 1];
        if (threadIdx.y == 0 && y > 0) // Vizinho de cima
            shared_plate[local_x][0] = d_plate[(y - 1) * n + x];
        if (threadIdx.y == BLOCK_SIZE - 1 && y < n - 1) // Vizinho de baixo
            shared_plate[local_x][BLOCK_SIZE + 1] = d_plate[(y + 1) * n + x];

        __syncthreads();

        // Atualizar a célula usando a equação de calor
        if (x > 0 && x < n - 1 && y > 0 && y < n - 1) {
            d_new_plate[y * n + x] = (shared_plate[local_x - 1][local_y] +
                                      shared_plate[local_x + 1][local_y] +
                                      shared_plate[local_x][local_y - 1] +
                                      shared_plate[local_x][local_y + 1]) / 4.0;
        }
    }
}

void initialize_plate(std::vector<float>& plate, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == 0) plate[i * n + j] = 100.0f; // Bordas superiores
            else plate[i * n + j] = 0.0f;         // Demais células
        }
    }
}

int main() {
    size_t size = N * N * sizeof(float);

    // Alocar memória na CPU
    std::vector<float> h_plate(N * N);
    initialize_plate(h_plate, N);

    // Alocar memória na GPU
    float* d_plate;
    float* d_new_plate;
    hipMalloc(&d_plate, size);
    hipMalloc(&d_new_plate, size);

    hipMemcpy(d_plate, h_plate.data(), size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(N / BLOCK_SIZE, N / BLOCK_SIZE);

    for (int t = 0; t < T; ++t) {
        heat_kernel<<<numBlocks, threadsPerBlock>>>(d_plate, d_new_plate, N);
        hipDeviceSynchronize();

        // Trocar os ponteiros das placas
        std::swap(d_plate, d_new_plate);
    }

    hipMemcpy(h_plate.data(), d_plate, size, hipMemcpyDeviceToHost);

    // Imprimir uma pequena seção da placa
    for (int i = 0; i < 10; ++i) {
        for (int j = 0; j < 10; ++j) {
            std::cout << std::setw(8) << std::setprecision(2) << h_plate[i * N + j] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_plate);
    hipFree(d_new_plate);

    return 0;
}
